#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

struct vec3 {
	float x, y, z;
};

__global__ void vectorAdd(struct vec3 *v1, struct vec3 *v2, struct vec3 *result) {
	int tid = threadIdx.x;
	/*	  1-1. write vector addition function						*/

	result->x = v1->x + v2->x;
	result->y = v1->y + v2->y;
	result->z = v1->z + v2->z;
}


int main( void )
{
	/*	  2-1. Check whether a proper device is mounted 			*/
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stdout, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
    }
    
	/*	  2-2. Declare Host and Device pointer variables			*/
    struct vec3 *a, *b, *c;
    struct vec3 *dev_a, *dev_b, *dev_c;

	/*    2-3. Allocate Host memory									*/
	a = (struct vec3*)malloc(sizeof(struct vec3));
	b = (struct vec3*)malloc(sizeof(struct vec3));
	c = (struct vec3*)malloc(sizeof(struct vec3));
    
	/*    2-4. Allocate Device memory								*/
	cudaStatus = hipMalloc((void**) &dev_a, sizeof(struct vec3));
	cudaStatus = hipMalloc((void**) &dev_b, sizeof(struct vec3));
	cudaStatus = hipMalloc((void**) &dev_c, sizeof(struct vec3));
    
	/*    2-5. Check that memory is allocated well on Device		*/
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    
	/*    2-6. Setup Input values to host array						*/
	a->x = 1;
	a->y = 2;
	a->z = 3;
	b->x = 10;
	b->y = 20;
	b->z = 30;

	/*    2-7. Copy memory for Input array from Host to Device		*/
    cudaStatus = hipMemcpy(dev_a, a, sizeof(struct vec3), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyHostToDevice a failed!");
	}
	
	cudaStatus = hipMemcpy(dev_b, b, sizeof(struct vec3), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyHostToDevice b failed!");
    }

	/*	  2-8. Call Kernel Function with <<<1, 1>>>					*/
	vectorAdd<<<1,1>>>(dev_a, dev_b, dev_c);

	/*    2-9. Copy memory for Result from Device to Host			*/
	cudaStatus = hipMemcpy(c, dev_c, sizeof(struct vec3), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyDeviceToHost failed!");
	}
	
	/*    2-10. Print Results										*/
	fprintf(stdout, "a: {x=%f, y=%f, z=%f}\n", a->x, a->y, a->z);
	fprintf(stdout, "b: {x=%f, y=%f, z=%f}\n", b->x, b->y, b->z);
	fprintf(stdout, "sum: {x=%f, y=%f, z=%f}\n", c->x, c->y, c->z);

	/*    2-11. Release Host and Device memory						*/
	free(a);
	free(b);
	free(c);
	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;

}

// nvcc [fileName] -o [outName]